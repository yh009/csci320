#include "hip/hip_runtime.h"
//Yuxuan Huang
#include <stdio.h>

__global__ void square(float * d_out, float * d_in){
    int idx = threadIdx.x + blockIdx.x*(2048/4);
    float f = d_in[idx];
    d_out[idx] = f * f;
}

int main(int argc, char ** argv) {
	const int ARRAY_SIZE;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
	const int BLOCK_NUM;
	const int BLOCK_SIZE=1024;
	const int BLOCK_SIZE_R;

	// taking user input
	printf("Please input an integer value: ");
	scanf("%d", &ARRAY_SIZE);

	// calculate parameters
	BLOCK_NUM = (int)ARRAY_SIZE/1024;
	BLOCK_SIZE_R = (int)ARRAY_SIZE % 1024;



	// generate the input array on the host
	float h_in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++) {
		h_in[i] = float(i);
	}
	float h_out[ARRAY_SIZE];

	// declare GPU memory pointers
	float * d_in;
	float * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	// transfer the array to the GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	// launch the kernel
	square<<<BLOCK_NUM, BLOCK_SIZE>>>(d_out, d_in);

	if 

	// copy back the result array to the CPU
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	// print out the resulting array
	for (int i =0; i < ARRAY_SIZE; i++) {
		printf("%f", h_out[i]);
		printf(((i % 4) != 3) ? "\t" : "\n");
	}

	hipFree(d_in);
	hipFree(d_out);

	hipError_t err;
   	if ( hipSuccess != (err = hipGetLastError()) ){
         fprintf(stderr, "CUDA error: %s\n", hipGetErrorString( err ) );
       	  exit(-2);
   	 }

	return 0;
}