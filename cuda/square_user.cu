//Yuxuan Huang

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square(float * d_out, float * d_in, int N){
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if (idx < N)
    {
    	float f = d_in[idx];
    	d_out[idx] = f * f;
    }
}

int main(int argc, char ** argv) {
	int ARRAY_SIZE;
	

	// taking user input
	printf("Please input an integer value: ");
	scanf("%d", &ARRAY_SIZE);

	int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);



	// generate the input array on the host
	float h_in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++) {
		h_in[i] = float(i);
	}
	float h_out[ARRAY_SIZE];

	// declare GPU memory pointers
	float * d_in;
	float * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	// transfer the array to the GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	// launch the kernel
	// Determine the number of blocks needed if ARRAY_SIZE is too large.
	int numBlocks = 1, threadsPerBlock = ARRAY_SIZE;
	if (ARRAY_SIZE > 1024)
	{
		numBlocks = ARRAY_SIZE/1024 + 1;
		threadsPerBlock = 1024;
		
	}
	square<<<numBlocks, threadsPerBlock>>>(d_out, d_in, ARRAY_SIZE);


	// copy back the result array to the CPU
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	// print out the resulting array
	for (int i =0; i < ARRAY_SIZE; i++) {
		printf("%f", h_out[i]);
		printf(((i % 4) != 3) ? "\t" : "\n");
	}

	hipFree(d_in);
	hipFree(d_out);

	hipError_t err;
   	if ( hipSuccess != (err = hipGetLastError()) ){
         fprintf(stderr, "CUDA error: %s\n", hipGetErrorString( err ) );
       	  exit(-2);
   	 }

	return 0;
}
